#include "hip/hip_runtime.h"

#include<bits/stdc++.h>
#include <numeric>
#include<math.h>
//#define 10 10

using namespace std;

__global__ void cuda_add(int *da,int *db,int *dc)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;	

	//dc[i] = da[i]=db[i]=0;
	dc[i] = da[i]+db[i];
	//printf("IN GLOBAL %d > %d %d %d \n",i,da[i],db[i],dc[i]);
}

int main()
{
	int a[10],b[10],c[10];
	cout<<"INITIALIZE ARRAY A";	
	
	for(int i=0;i<10;i++)
		{
		cin>>a[i];	
		}
		
	cout<<"INITIALIZE ARRAY B";	
	for(int i=0;i<10;i++)
		{
		cin>>b[i];	
		}
		
	int *da, *db, *dc;
	
	hipMalloc(&da,10*sizeof(int));
	hipMalloc(&db,10*sizeof(int));
	hipMalloc(&dc,10*sizeof(int));
	
	hipMemcpy(da,a,10*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(db,b,10*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dc,c,10*sizeof(int),hipMemcpyHostToDevice);

	cuda_add<<<2,(10/2)>>>(da,db,dc);

	hipMemcpy(c,dc,10*sizeof(int),hipMemcpyDeviceToHost);

	for(int i=0;i<10;i++)
		{
		cout<<c[i]<<"\t";	
		}
		
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	return 0;
}




